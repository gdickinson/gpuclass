#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>

/*
TODO: Head Comment

Copyright 2012 Guy Dickinson <guy.dickinson@nyu.edu>
*/

// Vanilla, sequential reduction on host
// This *would* have a divergence problem if it were multithreaded
int findMaxOnHost(int* A, int length) {
    for (int step = 1; step < length; step *= 2) {
        for (int i = 0; i < length; i += (2 * step)) {
            int candidate = A[i + step];
            if (A[i] < candidate) {
                A[i] = candidate;
            }
        }
    }
    return A[0];
}

__global__ void findMaxNaivelyKernel(int* A, int length) {
    unsigned int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    for (unsigned int step = 1; step < blockDim.x; step *= 2) {
        __syncthreads();
        if (tid % (2 * step) == 0) {
            int candidate = A[tid + step];
            if (A[tid] < candidate) {
                A[tid] = candidate;
            }
        }
        __syncthreads();
    }
    
}

// Naively find the maximum element, without taking into account thread
// diversion or memory efficiency
int cudaFindMaxOnDeviceNaively(int* A, int length) {
    int size = length * sizeof(int);
    int ret;
    int* hipArray;
    hipMalloc(&hipArray, size);
    hipMemcpy(hipArray, A, size, hipMemcpyHostToDevice);
    
    dim3 dimBlock(1024 % length);
    dim3 dimGrid(length / 1024, 1);
    
    findMaxNaivelyKernel<<<dimGrid, dimBlock>>>(hipArray, length);
    
    // Recover just the first element from the device to save time.
    hipMemcpy(&ret, hipArray, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(hipArray);
    return ret;
}




int* initializeArray(int length) {
    // Set up an array of ints of the right length
    void *ptr;
    ptr = malloc(length * sizeof(int));
    if (ptr == NULL) {
        // Handle allocation error
    }
    int* A = (int*) ptr;
    
    // Fill it with ints
    int j = 1;
    for (int i = 0; i < length; i++) {
        A[i] = j;
        j += 2;
    }
    return A;
}


void runTest(int length) {
    timeval serialStart, serialEnd;
    int* serialArr = initializeArray(length);
    int* naiveParallelArray = initializeArray(length);
    
    gettimeofday(&serialStart, NULL);
    int serialMax = findMaxOnHost(serialArr, length);
    gettimeofday(&serialEnd, NULL);
    free(serialArr);
    
    int expectedParallelMax = naiveParallelArray[length - 1];
    int naiveParallelMax =
        cudaFindMaxOnDeviceNaively(naiveParallelArray, length);
    
    
    
    // Make sure we actually found the max value
    if (serialMax == serialArr[length - 1]) {
        printf("Serial reduction passed check (expected %d, got %d)\n",
            serialMax, serialArr[length - 1]);
    } else {
        printf("Serial reduction failed! (expected %d, got %d)\n",
            serialMax, serialArr[length - 1]);
    }
    
    if (expectedParallelMax == naiveParallelArray[length - 1]) {
        printf("Naive Parallel reduction passed check (expected %d, got %d)\n",
            expectedParallelMax, naiveParallelArray[length - 1]);
    } else {
        printf("Naive Parallel reduction failed! (expected %d, got %d)\n",
            expectedParallelMax, naiveParallelArray[length - 1]);
    }
    
    double serialElapsedTime =
        (serialEnd.tv_sec - serialStart.tv_sec) * 1000.0;
    printf("Serial time: %g\n", serialElapsedTime); 
}

int main(void) {
    runTest(1024);
    return 0;
}